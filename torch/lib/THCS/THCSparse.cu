#include "THCSparse.h"

void THCudaSparse_Xcoo2csr(THCState *state, const int *coorowind, int64_t nnz, int64_t m, int *csrrowptr) {
  THAssertMsg((m <= INT_MAX) && (nnz <= INT_MAX),
    "hipsparseXcoo2csr only supports m, nnz with the bound [val] <= %d",
    INT_MAX);
  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  THCusparseCheck(hipsparseXcoo2csr(handle, coorowind, nnz, m, csrrowptr,
    TH_INDEX_BASE ? HIPSPARSE_INDEX_BASE_ONE : HIPSPARSE_INDEX_BASE_ZERO
  ));
}

hipsparseOperation_t convertTransToCusparseOperation(char trans) {
  if (trans == 't') return HIPSPARSE_OPERATION_TRANSPOSE;
  else if (trans == 'n') return HIPSPARSE_OPERATION_NON_TRANSPOSE;
  else if (trans == 'c') return HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  else {
    THError("trans must be one of: t, n, c");
    return HIPSPARSE_OPERATION_TRANSPOSE;
  }
}

void adjustLd(char transb, int64_t m, int64_t n, int64_t k, int64_t *ldb, int64_t *ldc)
{
  int transb_ = ((transb == 't') || (transb == 'T'));

  if(n == 1)
    *ldc = m;

  if(transb_)
  {
    if(k == 1)
      *ldb = n;
  }
  else
  {
    if(n == 1)
      *ldb = k;
  }
}

/* Level 3 */
void THCudaSparse_Scsrmm2(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, int64_t nnz, float alpha, float *csrvala, int *csrrowptra, int *csrcolinda, float *b, int64_t ldb, float beta, float *c, int64_t ldc)
{
  adjustLd(transb, m, n, k, &ldb, &ldc);
  hipsparseOperation_t opa = convertTransToCusparseOperation(transa);
  hipsparseOperation_t opb = convertTransToCusparseOperation(transb);

  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (nnz <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX),
    "hipsparseScsrmm2 only supports m, n, k, nnz, ldb, ldc with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_k = (int)k;
  int i_nnz = (int)nnz;
  int i_ldb = (int)ldb;
  int i_ldc = (int)ldc;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  hipsparseMatDescr_t desc;
  hipsparseCreateMatDescr(&desc);
#if TH_INDEX_BASE == 1
  hipsparseSetMatIndexBase(&desc, HIPSPARSE_INDEX_BASE_ONE);
#endif
  THCusparseCheck(hipsparseScsrmm2(handle, opa, opb, i_m, i_n, i_k, i_nnz, &alpha, desc, csrvala, csrrowptra, csrcolinda, b, i_ldb, &beta, c, i_ldc));
}

void THCudaSparse_Dcsrmm2(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, int64_t nnz, double alpha, double *csrvala, int *csrrowptra, int *csrcolinda, double *b, int64_t ldb, double beta, double *c, int64_t ldc)
{
  adjustLd(transb, m, n, k, &ldb, &ldc);
  hipsparseOperation_t opa = convertTransToCusparseOperation(transa);
  hipsparseOperation_t opb = convertTransToCusparseOperation(transb);

  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (nnz <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX),
    "hipsparseDcsrmm2 only supports m, n, k, nnz, ldb, ldc with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_k = (int)k;
  int i_nnz = (int)nnz;
  int i_ldb = (int)ldb;
  int i_ldc = (int)ldc;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  hipsparseMatDescr_t desc;
  hipsparseCreateMatDescr(&desc);
#if TH_INDEX_BASE == 1
  hipsparseSetMatIndexBase(&desc, HIPSPARSE_INDEX_BASE_ONE);
#endif
  THCusparseCheck(hipsparseDcsrmm2(handle, opa, opb, i_m, i_n, i_k, i_nnz, &alpha, desc, csrvala, csrrowptra, csrcolinda, b, i_ldb, &beta, c, i_ldc));
}

void THCudaSparse_Scsrgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, int64_t nnza, float *csrvala, int *csrrowptra, int *csrcolinda, int64_t nnzb, float *csrvalb, int *csrrowptrb, int *csrcolindb, int64_t *nnzc, float *csrvalc, int *csrrowptrc, int *csrcolindc)
{

  hipsparseOperation_t opa = convertTransToCusparseOperation(transa);
  hipsparseOperation_t opb = convertTransToCusparseOperation(transb);

  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (nnza <= INT_MAX) && (nnzb <= INT_MAX),
    "hipsparseScsrgemm only supports m, n, k, nnzA, nnzB, and nnzC with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_k = (int)k;
  int i_nnza = (int)nnza;
  int i_nnzb = (int)nnzb;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));

  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  hipsparseMatDescr_t descrB;
  hipsparseCreateMatDescr(&descrB);
  hipsparseMatDescr_t descrC;
  hipsparseCreateMatDescr(&descrC);

#if TH_INDEX_BASE == 1
  hipsparseSetMatIndexBase(&descrA, HIPSPARSE_INDEX_BASE_ONE);
  hipsparseSetMatIndexBase(&descrB, HIPSPARSE_INDEX_BASE_ONE);
  hipsparseSetMatIndexBase(&descrC, HIPSPARSE_INDEX_BASE_ONE);
#endif

	int baseC;
	int *nnzTotalDevHostPtr = 0;

	THCudaCheck(THCudaMalloc(state, (void**)(&csrrowptrc), m+1));

  THCusparseCheck(hipsparseXcsrgemmNnz(handle, opa, opb, i_m, i_n, i_k,
          descrA, i_nnza, csrrowptra, csrcolinda,
          descrB, i_nnzb, csrrowptrb, csrcolindb,
          descrC, csrrowptrc, nnzTotalDevHostPtr));

  *nnzc = (int64_t)(*nnzTotalDevHostPtr);

	THCudaCheck(THCudaMalloc(state, (void**)(&csrcolindc), *nnzc));
	THCudaCheck(THCudaMalloc(state, (void**)(&csrvalc), *nnzc));

  THCusparseCheck(hipsparseScsrgemm(handle, opa, opb, i_m, i_n, i_k,
        descrA, i_nnza, csrvala, csrrowptra, csrcolinda,
        descrB, i_nnzb, csrvalb, csrrowptrb, csrcolindb,
        descrC, csrvalc, csrrowptrc, csrcolindc));
}



void THCudaSparse_Dcsrgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, int64_t nnza, double *csrvala, int *csrrowptra, int *csrcolinda, int64_t nnzb, double *csrvalb, int *csrrowptrb, int *csrcolindb, int64_t *nnzc, double *csrvalc, int *csrrowptrc, int *csrcolindc)
{

  hipsparseOperation_t opa = convertTransToCusparseOperation(transa);
  hipsparseOperation_t opb = convertTransToCusparseOperation(transb);

  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (nnza <= INT_MAX) && (nnzb <= INT_MAX),
    "hipsparseScsrgemm only supports m, n, k, nnzA, nnzB, and nnzC with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_k = (int)k;
  int i_nnza = (int)nnza;
  int i_nnzb = (int)nnzb;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));

  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  hipsparseMatDescr_t descrB;
  hipsparseCreateMatDescr(&descrB);
  hipsparseMatDescr_t descrC;
  hipsparseCreateMatDescr(&descrC);

#if TH_INDEX_BASE == 1
  hipsparseSetMatIndexBase(&descrA, HIPSPARSE_INDEX_BASE_ONE);
  hipsparseSetMatIndexBase(&descrB, HIPSPARSE_INDEX_BASE_ONE);
  hipsparseSetMatIndexBase(&descrC, HIPSPARSE_INDEX_BASE_ONE);
#endif

	int baseC, nnzC = 0; // nnzTotalDevHostPtr points to host memory int *nnzTotalDevHostPtr = &nnzC;
	int *nnzTotalDevHostPtr = &nnzC;

	printf("m: %d\n", m);
	printf("nnza: %ld\n", i_nnza);
	printf("nnzb: %ld\n", i_nnzb);

	THCudaCheck(THCudaMalloc(state, (void**)(&csrrowptrc), m+1));

	printf("post pointer \n");

  printf("m: %d \n", i_m);
  printf("n: %d \n", i_n);
  printf("k: %d \n", i_k);
  printf("nnza: %d \n", i_nnza);
  printf("nnzb: %d \n", i_nnzb);

  THCusparseCheck(hipsparseXcsrgemmNnz(handle, opa, opb, i_m, i_n, i_k,
          descrA, i_nnza, csrrowptra, csrcolinda,
          descrB, i_nnzb, csrrowptrb, csrcolindb,
          descrC, csrrowptrc, nnzTotalDevHostPtr));

  if (NULL != nnzTotalDevHostPtr)
  {
			printf("post nnzc %d \n", nnzC);
      nnzC = *nnzTotalDevHostPtr;
			printf("assigned!\n");
			printf("post nnzc %d \n", nnzC);
	}
	else
	{

		printf("post nnzc %d \n", *nnzTotalDevHostPtr);
		printf("post nnzc %d \n", nnzC);
	}


  *nnzc = int64_t(*nnzTotalDevHostPtr);

  printf("nnzc2: %ld", nnzc);

	THCudaCheck(THCudaMalloc(state, (void**)(&csrcolindc), nnzC));
	THCudaCheck(THCudaMalloc(state, (void**)(&csrvalc), nnzC));

	printf("post alloc \n");

  THCusparseCheck(hipsparseDcsrgemm(handle, opa, opb, i_m, i_n, i_k,
        descrA, i_nnza, csrvala, csrrowptra, csrcolinda,
        descrB, i_nnzb, csrvalb, csrrowptrb, csrcolindb,
        descrC, csrvalc, csrrowptrc, csrcolindc));

	printf("post gemm \n");
}

/* format conversion */
void THCudaSparse_CreateIdentityPermutation(THCState *state, int64_t nnz, int *P) {
  THAssertMsg((nnz <= INT_MAX),
    "Xcsrsort_bufferSizeExt only supports m, n, nnz with the bound [val] <= %d",
    INT_MAX);
  int i_nnz = (int)nnz;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  hipsparseCreateIdentityPermutation(handle, i_nnz, P);
}

void THCudaSparse_Xcsrsort_bufferSizeExt(THCState *state, int64_t m, int64_t n, int64_t nnz, const int *csrRowPtr, const int *csrColInd, size_t *pBufferSizeInBytes)
{
  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (nnz <= INT_MAX),
    "Xcsrsort_bufferSizeExt only supports m, n, nnz with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_nnz = (int)nnz;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  THCusparseCheck(hipsparseXcsrsort_bufferSizeExt(handle, i_m, i_n, i_nnz, csrRowPtr, csrColInd, pBufferSizeInBytes));
}

void THCudaSparse_Xcsrsort(THCState *state, int64_t m, int64_t n, int64_t nnz, const int *csrRowPtr, int *csrColInd, int *P, void *pBuffer)
{
  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (nnz <= INT_MAX),
    "Xcsrsort only supports m, n, nnz with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_nnz = (int)nnz;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  hipsparseMatDescr_t desc;
  hipsparseCreateMatDescr(&desc);
#if TH_INDEX_BASE == 1
  hipsparseSetMatIndexBase(&desc, HIPSPARSE_INDEX_BASE_ONE);
#endif
  THCusparseCheck(hipsparseXcsrsort(handle, i_m, i_n, i_nnz, desc, csrRowPtr, csrColInd, P, pBuffer));
}

void THCudaSparse_Xcoosort_bufferSizeExt(THCState *state, int64_t m, int64_t n, int64_t nnz, const int *cooRows, const int *cooCols, size_t *pBufferSizeInBytes)
{
  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (nnz <= INT_MAX),
    "Xcoosort_bufferSizeExt only supports m, n, nnz with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_nnz = (int)nnz;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  THCusparseCheck(hipsparseXcoosort_bufferSizeExt(handle, i_m, i_n, i_nnz, cooRows, cooCols, pBufferSizeInBytes));
}

void THCudaSparse_XcoosortByRow(THCState *state, int64_t m, int64_t n, int64_t nnz, int *cooRows, int *cooCols, int *P, void *pBuffer)
{
  THAssertMsg((m <= INT_MAX) && (n <= INT_MAX) && (nnz <= INT_MAX),
    "XcoosortByRow only supports m, n, nnz with the bound [val] <= %d",
    INT_MAX);
  int i_m = (int)m;
  int i_n = (int)n;
  int i_nnz = (int)nnz;

  hipsparseHandle_t handle = THCState_getCurrentSparseHandle(state);
  hipsparseSetStream(handle, THCState_getCurrentStream(state));
  THCusparseCheck(hipsparseXcoosortByRow(handle, i_m, i_n, i_nnz, cooRows, cooCols, P, pBuffer));
}
